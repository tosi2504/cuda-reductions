#include "../headers/kernels.cuh"
#include <iostream>
#include <fstream>
#include <random>
#include <chrono>
#include <stdint.h>

double timeKernel4(const int n) {
    // set the datatype
    typedef int datatype;

	// set the parallelisation parameters
    const int numThreadsPerBlock = 256;
	const int N = ROUND_UP(n, 2*numThreadsPerBlock); // NOTE: for kernels 0,1,2 dont need the 2*...

	// allocate memory on host for input and output
	datatype * h_in = (datatype*)malloc(sizeof(datatype)*N);
    datatype * h_out = (datatype*)malloc(sizeof(datatype));

	// fill array with random numbers
	std::mt19937 rng(106);
	//std::uniform_real_distribution<double> dist(0,1); 
	std::uniform_int_distribution<> dist(0,10); 
	for (int i = 0; i < n; ++i) {
		h_in[i] = dist(rng);
		//h_in[i] = 1;
	}
    // zeropadding
	for (int i = n; i < N; ++i) {
		h_in[i] = 0;
	}

	// allocate and populate memory on the device
	datatype * d_in, * d_temp1, * d_temp2, * d_out;
	CCE(hipMalloc(&d_in, sizeof(datatype)*N));
	CCE(hipMemcpy(d_in, h_in, sizeof(datatype)*N, hipMemcpyHostToDevice));
    int numBlocks1 = DIV_UP(N, numThreadsPerBlock);
	CCE(hipMalloc(&d_temp1, sizeof(datatype)*ROUND_UP(numBlocks1, 2*numThreadsPerBlock))); // does not need to be initialized
    // NOTE: 2* is not required for kernels 0,1,2
    int numBlocks2 = DIV_UP(numBlocks1, numThreadsPerBlock);
	CCE(hipMalloc(&d_temp2, sizeof(datatype)*ROUND_UP(numBlocks2, 2*numThreadsPerBlock))); // does not need to be initialized
    // NOTE: 2* is not required for kernels 0,1,2

	// benchmark the kernel
    unsigned int numCycles = 1<<8;

    auto start = std::chrono::high_resolution_clock::now();
    for (unsigned int i = 0; i < numCycles; ++i) {
        wrapperKer_4 < datatype > (d_in, d_temp1, d_temp2, &d_out, N, numThreadsPerBlock);
        hipDeviceSynchronize();
    }
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop-start);
	CCE(hipMemcpy(h_out, d_out, sizeof(datatype), hipMemcpyDeviceToHost));
    CCEL();

    hipFree(d_in);
    CCEL();
    hipFree(d_out);
    CCEL();
    //hipFree(d_temp1);
    CCEL();
    //hipFree(d_temp2);
    CCEL();
    free(h_in);
    free(h_out);

    return duration.count()/(double)numCycles;
}


int main () {
    std::ofstream file("data.txt");
    // time the kernel
    int N = 1024;
    for (int i = 0; i < 21; i++) {
        std::cout << N << " " << timeKernel4(N) << std::endl;
        file << N << " " << timeKernel4(N) << "\n";
        N *= 2;
    }
    file.close();
}

